#include "hip/hip_runtime.h"
:insert
/*
 * File Name     :
 * Purpose       :
 * Creation Date :
 * Last Modified : Tue 01 Nov 2016 05:24:13 PM CET
 * Created By    :
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <signal.h>
#include <fcntl.h>
#include <sys/file.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <netdb.h>
#include <hip/hip_runtime.h>

// hello from GPU kernel
__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU! \n");
}

int main (int argc, char *argv[])
{
    printf("Step one\n");
    printf("???\n");
    printf("Profit\n");

    helloFromGPU <<<1, 10>>>();
   
    // release the device
    hipDeviceReset();
    return 0;
}
